#include <stdio.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

__device__ int counter = 100;

__global__ void increment()
{
    counter++;
}

void checkCuda(hipError_t result, const char *msg) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s - %s\n", msg, hipGetErrorString(result));
        exit(1);
    }
}

int main(void)
{
    // Initialize CUDA
    checkCuda(hipFree(0), "Initializing CUDA");

    // Initialize counter to 100 on the device
    int initialCounter = 100;
    checkCuda(hipMemcpyToSymbol(HIP_SYMBOL(counter), &initialCounter, sizeof(int)), "Initializing counter");

    while (true) {
        int hCounter = 0;

        // Launch the increment kernel
        increment<<<1, 1>>>();
        checkCuda(hipDeviceSynchronize(), "Kernel execution");

        // Copy the counter from device to host
        checkCuda(hipMemcpyFromSymbol(&hCounter, HIP_SYMBOL(counter), sizeof(counter)), "Copying counter to host");

        // Print the current counter value
        printf("%d\n", hCounter);

        // Wait for 1 second
        sleep(1);
    }

    return 0;
}

